
#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include <stdlib.h>
#include <iostream>
#include <fstream>
#include <iomanip>
#include <cstring>
#include <string>
#include <vector>
#include <type_traits>
#include <Windows.h>
#include <hipblas.h>

#define cudacall(call)                                                                                                          \
    do                                                                                                                          \
    {                                                                                                                           \
        hipError_t err = (call);                                                                                               \
        if(hipSuccess != err)                                                                                                  \
        {                                                                                                                       \
            fprintf(stderr,"CUDA Error:\nFile = %s\nLine = %d\nReason = %s\n", __FILE__, __LINE__, hipGetErrorString(err));    \
            hipDeviceReset();                                                                                                  \
            exit(EXIT_FAILURE);                                                                                                 \
        }                                                                                                                       \
    }                                                                                                                           \
    while (0)

#define cublascall(call)                                                                                        \
    do                                                                                                          \
    {                                                                                                           \
        hipblasStatus_t status = (call);                                                                         \
        if(HIPBLAS_STATUS_SUCCESS != status)                                                                     \
        {                                                                                                       \
            fprintf(stderr,"CUBLAS Error:\nFile = %s\nLine = %d\nCode = %d\n", __FILE__, __LINE__, status);     \
            hipDeviceReset();                                                                                  \
            exit(EXIT_FAILURE);                                                                                 \
        }                                                                                                       \
                                                                                                                \
    }                                                                                                           \
    while(0)

#define PREC_SAVE 10

template <class T>
void write_1D(const std::string fname, T* ptr, size_t sz0)
{
	std::ofstream ofs;
	ofs.open(fname, std::ofstream::out);
	ofs.setf(std::ios::scientific & std::ios::floatfield);
	ofs.precision(PREC_SAVE);

	for (auto i = 0; i != sz0; ++i)
		ofs << ptr[i] << "\n";
	ofs.close();
}

template <class T>
void write_2D(const std::string fname, T** ptr, size_t sz0, size_t sz1)
{
	std::ofstream ofs;
	ofs.open(fname, std::ofstream::out);
	ofs.setf(std::ios::scientific & std::ios::floatfield);
	ofs.precision(PREC_SAVE);

	for (auto i = 0; i != sz0; ++i)
		for (auto j = 0; j != sz1; ++j)
			ofs << ptr[i][j] << "\n";
	ofs.close();
}

template <class T>
void read_1D(const std::string fname, T* ptr, size_t sz0)
{
	std::ifstream ifs;
	ifs.open(fname, std::ifstream::in);

	for (auto i = 0; i != sz0; ++i)
		ifs >> ptr[i];
	ifs.close();
}

template <class T>
void read_2D(const std::string fname, T** ptr, size_t sz0, size_t sz1)
{
	std::ifstream ifs;
	ifs.open(fname, std::ifstream::in);

	for (auto i = 0; i != sz0; ++i)
		for (auto j = 0; j != sz1; ++j)
			ifs >> ptr[i][j];
	ifs.close();
}

template <class T>
void setHankelMatrix(T* __restrict h_A, const int n) {

	T* h_atemp = (T*)malloc((2 * n - 1) * sizeof(T));

	// --- Initialize random seed
	srand(time(NULL));

	// --- Generate random numbers
	for (int k = 0; k < 2 * n - 1; k++) h_atemp[k] = static_cast<T>(rand());

	// --- Fill the Hankel matrix. The Hankel matrix is symmetric, so filling by row or column is equivalent.
	for (int i = 0; i < n; i++)
		for (int j = 0; j < n; j++)
			h_A[i * n + j] = h_atemp[(i + 1) + (j + 1) - 2];

	free(h_atemp);

}

template <class T>
void invert(T** src, T** dst, int n, int batchSize, LARGE_INTEGER& pc_diff_inner)
{
	LARGE_INTEGER pc_start_inner, pc_finish_inner;

	hipblasHandle_t handle;
	cublascall(hipblasCreate(&handle));

	int* P, * INFO;

	cudacall(hipMalloc(&P, n * batchSize * sizeof(int)));
	cudacall(hipMalloc(&INFO, batchSize * sizeof(int)));

	int lda = n;

	//int INFOh[batchSize];
	int* INFOh = new int[batchSize];

	T** A = (T**)malloc(batchSize * sizeof(T*));
	T** A_d, * A_dflat;
	cudacall(hipMalloc(&A_d, batchSize * sizeof(T*)));
	cudacall(hipMalloc(&A_dflat, n * n * batchSize * sizeof(T)));
	A[0] = A_dflat;
	for (int i = 1; i < batchSize; i++)
		A[i] = A[i - 1] + (n * n);
	cudacall(hipMemcpy(A_d, A, batchSize * sizeof(T*), hipMemcpyHostToDevice));
	for (int i = 0; i < batchSize; i++)
		cudacall(hipMemcpy(A_dflat + (i * n * n), src[i], n * n * sizeof(T), hipMemcpyHostToDevice));

	T** C = (T**)malloc(batchSize * sizeof(T*));
	T** C_d, * C_dflat;
	cudacall(hipMalloc(&C_d, batchSize * sizeof(T*)));
	cudacall(hipMalloc(&C_dflat, n * n * batchSize * sizeof(T)));
	C[0] = C_dflat;
	for (int i = 1; i < batchSize; i++)
		C[i] = C[i - 1] + (n * n);
	cudacall(hipMemcpy(C_d, C, batchSize * sizeof(T*), hipMemcpyHostToDevice));

	hipDeviceSynchronize();
	QueryPerformanceCounter(&pc_start_inner);

	if (std::is_same<T, double>::value)
		cublascall(hipblasDgetrfBatched(handle, n, (double**)A_d, lda, P, INFO, batchSize));
	else if (std::is_same<T, float>::value)
		cublascall(hipblasSgetrfBatched(handle, n, (float**)A_d, lda, P, INFO, batchSize));

	//cudacall(hipMemcpy(INFOh, INFO, batchSize * sizeof(int), hipMemcpyDeviceToHost));
	//for (int i = 0; i < batchSize; i++)
	//	if (INFOh[i] != 0)
	//	{
	//		fprintf(stderr, "Factorization of matrix %d Failed: Matrix may be singular\n", i);
	//		hipDeviceReset();
	//		exit(EXIT_FAILURE);
	//	}

	if (std::is_same<T, double>::value)
		cublascall(hipblasDgetriBatched(handle, n, (const double**)A_d, lda, P, (double**)C_d, lda, INFO, batchSize));
	else if (std::is_same<T, float>::value)
		cublascall(hipblasSgetriBatched(handle, n, (const float**)A_d, lda, P, (float**)C_d, lda, INFO, batchSize));

	hipDeviceSynchronize();
	QueryPerformanceCounter(&pc_finish_inner);
	pc_diff_inner.QuadPart += pc_finish_inner.QuadPart - pc_start_inner.QuadPart;

	//cudacall(hipMemcpy(INFOh, INFO, batchSize * sizeof(int), hipMemcpyDeviceToHost));
	//for (int i = 0; i < batchSize; i++)
	//	if (INFOh[i] != 0)
	//	{
	//		fprintf(stderr, "Inversion of matrix %d Failed: Matrix may be singular\n", i);
	//		hipDeviceReset();
	//		exit(EXIT_FAILURE);
	//	}
	for (int i = 0; i < batchSize; i++)
		cudacall(hipMemcpy(dst[i], C_dflat + (i * n * n), n * n * sizeof(T), hipMemcpyDeviceToHost));

	hipFree(A_d); hipFree(A_dflat); free(A);
	hipFree(C_d); hipFree(C_dflat); free(C);
	hipFree(P); hipFree(INFO); hipblasDestroy(handle);
	delete[] INFOh;
}

template <class T>
void test_invert(const int n, const int mybatch, LARGE_INTEGER& pc_diff_inner, T null, const size_t testnum)
{
	T** results = (T**)malloc(mybatch * sizeof(T*));
	for (auto i = 0; i != mybatch; ++i)
		results[i] = (T*)malloc(n * n * sizeof(T));

	T** inputs = (T**)malloc(mybatch * sizeof(T*));
	for (auto i = 0; i != mybatch; ++i)
		inputs[i] = (T*)malloc(n * n * sizeof(T));
	//for (auto i = 0; i != mybatch; ++i)
	//	read_1D("A" + std::to_string(n) + ".txt", inputs[i], n * n);
	for (auto i = 0; i != mybatch; ++i)
		setHankelMatrix(inputs[i], n);

	for (auto i = 0; i!=testnum; ++i)
		invert(inputs, results, n, mybatch, pc_diff_inner);

	//write_1D("iA.txt", results[0], n * n);
	for (auto i = 0; i != mybatch; ++i)
	{
		free(inputs[i]);
		free(results[i]);
	}
}

template <class T>
void test_invert_onetypeall(const std::vector<int>& v_n, const std::vector<int>& v_mybatch, std::vector<std::vector<LARGE_INTEGER>>& v_pc_diff_inner, LARGE_INTEGER& pf, T t_null, const size_t testnum)
{
	for (auto& tmp0 : v_pc_diff_inner)
		for (auto& tmp1 : tmp0)
			ZeroMemory(&tmp1, sizeof(LARGE_INTEGER));
	for (auto i = 0; i != v_n.size(); ++i)
		for (auto j = 0; j != v_mybatch.size(); ++j)
			test_invert(v_n[i], v_mybatch[j], v_pc_diff_inner[i][j], t_null, testnum);
	std::cout << "Type=" << typeid(t_null).name() << std::endl;
	for (auto i = 0; i != v_n.size(); ++i)
		for (auto j = 0; j != v_mybatch.size(); ++j)
			std::cout << "n = " << v_n[i] << ", mybatch = " << v_mybatch[j] << ", time = " << (v_pc_diff_inner[i][j].QuadPart) * 1000000 / pf.QuadPart / testnum << " microsecond." << std::endl;
	std::cout << std::endl;
}

int main()
{
	size_t testnum = 10;

	std::cout << "Test: Time comsuming of LU decomposition matrix inversion with CUDA." << std::endl << std::endl;

	int nDevices;
	hipGetDeviceCount(&nDevices);
	for (int i = 0; i < nDevices; i++)
	{
		hipDeviceProp_t prop;
		hipGetDeviceProperties(&prop, i);
		printf("Device Number: %d\n", i);
		printf("  Device name: %s\n", prop.name);
		printf("  Clock Rate (KHz): %d\n",
			prop.clockRate);
		printf("  Memory Clock Rate (KHz): %d\n",
			prop.memoryClockRate);
		printf("  Memory Bus Width (bits): %d\n",
			prop.memoryBusWidth);
		printf("  Peak Memory Bandwidth (GB/s): %f\n\n",
			2.0 * prop.memoryClockRate * (prop.memoryBusWidth / 8) / 1.0e6);
	}

	std::vector<int> v_n;
	v_n.push_back(16);
	v_n.push_back(32);
	v_n.push_back(64);
	v_n.push_back(128);
	v_n.push_back(256);
	v_n.push_back(512);
	v_n.push_back(1024);

	std::vector<int> v_mybatch;
	v_mybatch.push_back(1);

	LARGE_INTEGER pf;
	QueryPerformanceFrequency(&pf);

	std::vector<std::vector<LARGE_INTEGER>> v_pc_diff_inner;
	v_pc_diff_inner.resize(v_n.size());
	for (auto& tmp0 : v_pc_diff_inner)
		tmp0.resize(v_mybatch.size());

	double null_d;
	float null_f;
	test_invert_onetypeall(v_n, v_mybatch, v_pc_diff_inner, pf, null_d, testnum);
	test_invert_onetypeall(v_n, v_mybatch, v_pc_diff_inner, pf, null_f, testnum);

	return 0;
}