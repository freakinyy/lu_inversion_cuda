
#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include <stdlib.h>
#include <iostream>
#include <fstream>
#include <iomanip>
#include <cstring>
#include <string>
#include <vector>
#include <type_traits>
#include <Windows.h>
#include <hipblas.h>

#define cudacall(call)                                                                                                          \
    do                                                                                                                          \
    {                                                                                                                           \
        hipError_t err = (call);                                                                                               \
        if(hipSuccess != err)                                                                                                  \
        {                                                                                                                       \
            fprintf(stderr,"CUDA Error:\nFile = %s\nLine = %d\nReason = %s\n", __FILE__, __LINE__, hipGetErrorString(err));    \
            hipDeviceReset();                                                                                                  \
            exit(EXIT_FAILURE);                                                                                                 \
        }                                                                                                                       \
    }                                                                                                                           \
    while (0)

#define cublascall(call)                                                                                        \
    do                                                                                                          \
    {                                                                                                           \
        hipblasStatus_t status = (call);                                                                         \
        if(HIPBLAS_STATUS_SUCCESS != status)                                                                     \
        {                                                                                                       \
            fprintf(stderr,"CUBLAS Error:\nFile = %s\nLine = %d\nCode = %d\n", __FILE__, __LINE__, status);     \
            hipDeviceReset();                                                                                  \
            exit(EXIT_FAILURE);                                                                                 \
        }                                                                                                       \
                                                                                                                \
    }                                                                                                           \
    while(0)

#define PREC_SAVE 10

class TestItem
{
public:
	TestItem(int n, int mybatch = 1, std::string type = "double", bool decomponly = false, size_t testnum = 100) :n(n), mybatch(mybatch), type(type), decomponly(decomponly), testnum(testnum) {};
	~TestItem() {};
	int n;
	int mybatch;
	std::string type;
	bool decomponly;
	size_t testnum;
};

const std::vector<TestItem> LoadTestConfig(const std::string& fname)
{
	std::vector<TestItem> v_ti;
	int n;
	int mybatch;
	std::string type;
	std::string str_decomponly;
	bool decomponly;
	size_t testnum;

	std::ifstream ifs;
	ifs.open(fname);
	if (!ifs.is_open())
	{
		std::cout << "Fail to open file: \"" << fname << "\"" << std::endl;
		return v_ti;
	}
	while (!ifs.eof())
	{
		ifs >> n;
		ifs >> mybatch;
		ifs >> type;
		ifs >> str_decomponly;
		if (str_decomponly == "0")
			decomponly = false;
		else
			decomponly = true;
		ifs >> testnum;
		v_ti.push_back(TestItem(n, mybatch, type, decomponly, testnum));
	}
	return v_ti;
}

template <class T>
void write_1D(const std::string fname, T* ptr, size_t sz0)
{
	std::ofstream ofs;
	ofs.open(fname, std::ofstream::out);
	ofs.setf(std::ios::scientific & std::ios::floatfield);
	ofs.precision(PREC_SAVE);

	for (auto i = 0; i != sz0; ++i)
		ofs << ptr[i] << "\n";
	ofs.close();
}

template <class T>
void write_2D(const std::string fname, T** ptr, size_t sz0, size_t sz1)
{
	std::ofstream ofs;
	ofs.open(fname, std::ofstream::out);
	ofs.setf(std::ios::scientific & std::ios::floatfield);
	ofs.precision(PREC_SAVE);

	for (auto i = 0; i != sz0; ++i)
		for (auto j = 0; j != sz1; ++j)
			ofs << ptr[i][j] << "\n";
	ofs.close();
}

template <class T>
void read_1D(const std::string fname, T* ptr, size_t sz0)
{
	std::ifstream ifs;
	ifs.open(fname, std::ifstream::in);

	for (auto i = 0; i != sz0; ++i)
		ifs >> ptr[i];
	ifs.close();
}

template <class T>
void read_2D(const std::string fname, T** ptr, size_t sz0, size_t sz1)
{
	std::ifstream ifs;
	ifs.open(fname, std::ifstream::in);

	for (auto i = 0; i != sz0; ++i)
		for (auto j = 0; j != sz1; ++j)
			ifs >> ptr[i][j];
	ifs.close();
}

template <class T>
void setHankelMatrix(T* __restrict h_A, const int n) {

	T* h_atemp = (T*)malloc((2 * n - 1) * sizeof(T));

	// --- Initialize random seed
	srand(time(NULL));

	// --- Generate random numbers
	for (int k = 0; k < 2 * n - 1; k++) h_atemp[k] = static_cast<T>(rand());

	// --- Fill the Hankel matrix. The Hankel matrix is symmetric, so filling by row or column is equivalent.
	for (int i = 0; i < n; i++)
		for (int j = 0; j < n; j++)
			h_A[i * n + j] = h_atemp[(i + 1) + (j + 1) - 2];

	free(h_atemp);

}

template <class T>
void invert(T** src, T** dst, int n, int batchSize, LARGE_INTEGER& pc_diff_inner)
{
	LARGE_INTEGER pc_start_inner, pc_finish_inner;

	hipblasHandle_t handle;
	cublascall(hipblasCreate(&handle));

	int* P, * INFO;

	cudacall(hipMalloc(&P, n * batchSize * sizeof(int)));
	cudacall(hipMalloc(&INFO, batchSize * sizeof(int)));

	int lda = n;

	//int INFOh[batchSize];
	int* INFOh = new int[batchSize];

	T** A = (T**)malloc(batchSize * sizeof(T*));
	T** A_d, * A_dflat;
	cudacall(hipMalloc(&A_d, batchSize * sizeof(T*)));
	cudacall(hipMalloc(&A_dflat, n * n * batchSize * sizeof(T)));
	A[0] = A_dflat;
	for (int i = 1; i < batchSize; i++)
		A[i] = A[i - 1] + (n * n);
	cudacall(hipMemcpy(A_d, A, batchSize * sizeof(T*), hipMemcpyHostToDevice));
	for (int i = 0; i < batchSize; i++)
		cudacall(hipMemcpy(A_dflat + (i * n * n), src[i], n * n * sizeof(T), hipMemcpyHostToDevice));

	T** C = (T**)malloc(batchSize * sizeof(T*));
	T** C_d, * C_dflat;
	cudacall(hipMalloc(&C_d, batchSize * sizeof(T*)));
	cudacall(hipMalloc(&C_dflat, n * n * batchSize * sizeof(T)));
	C[0] = C_dflat;
	for (int i = 1; i < batchSize; i++)
		C[i] = C[i - 1] + (n * n);
	cudacall(hipMemcpy(C_d, C, batchSize * sizeof(T*), hipMemcpyHostToDevice));

	hipDeviceSynchronize();
	QueryPerformanceCounter(&pc_start_inner);

	if (std::is_same<T, double>::value)
		cublascall(hipblasDgetrfBatched(handle, n, (double**)A_d, lda, P, INFO, batchSize));
	else if (std::is_same<T, float>::value)
		cublascall(hipblasSgetrfBatched(handle, n, (float**)A_d, lda, P, INFO, batchSize));

	//cudacall(hipMemcpy(INFOh, INFO, batchSize * sizeof(int), hipMemcpyDeviceToHost));
	//for (int i = 0; i < batchSize; i++)
	//	if (INFOh[i] != 0)
	//	{
	//		fprintf(stderr, "Factorization of matrix %d Failed: Matrix may be singular\n", i);
	//		hipDeviceReset();
	//		exit(EXIT_FAILURE);
	//	}

	if (std::is_same<T, double>::value)
		cublascall(hipblasDgetriBatched(handle, n, (const double**)A_d, lda, P, (double**)C_d, lda, INFO, batchSize));
	else if (std::is_same<T, float>::value)
		cublascall(hipblasSgetriBatched(handle, n, (const float**)A_d, lda, P, (float**)C_d, lda, INFO, batchSize));

	hipDeviceSynchronize();
	QueryPerformanceCounter(&pc_finish_inner);
	pc_diff_inner.QuadPart += pc_finish_inner.QuadPart - pc_start_inner.QuadPart;

	//cudacall(hipMemcpy(INFOh, INFO, batchSize * sizeof(int), hipMemcpyDeviceToHost));
	//for (int i = 0; i < batchSize; i++)
	//	if (INFOh[i] != 0)
	//	{
	//		fprintf(stderr, "Inversion of matrix %d Failed: Matrix may be singular\n", i);
	//		hipDeviceReset();
	//		exit(EXIT_FAILURE);
	//	}
	for (int i = 0; i < batchSize; i++)
		cudacall(hipMemcpy(dst[i], C_dflat + (i * n * n), n * n * sizeof(T), hipMemcpyDeviceToHost));

	hipFree(A_d); hipFree(A_dflat); free(A);
	hipFree(C_d); hipFree(C_dflat); free(C);
	hipFree(P); hipFree(INFO); hipblasDestroy(handle);
	delete[] INFOh;
}

template <class T>
void decomposition_only(T** src, T** dst, int n, int batchSize, LARGE_INTEGER& pc_diff_inner)
{
	LARGE_INTEGER pc_start_inner, pc_finish_inner;

	hipblasHandle_t handle;
	cublascall(hipblasCreate(&handle));

	int* P, * INFO;

	cudacall(hipMalloc(&P, n * batchSize * sizeof(int)));
	cudacall(hipMalloc(&INFO, batchSize * sizeof(int)));

	int lda = n;

	//int INFOh[batchSize];
	int* INFOh = new int[batchSize];

	T** A = (T**)malloc(batchSize * sizeof(T*));
	T** A_d, * A_dflat;
	cudacall(hipMalloc(&A_d, batchSize * sizeof(T*)));
	cudacall(hipMalloc(&A_dflat, n * n * batchSize * sizeof(T)));
	A[0] = A_dflat;
	for (int i = 1; i < batchSize; i++)
		A[i] = A[i - 1] + (n * n);
	cudacall(hipMemcpy(A_d, A, batchSize * sizeof(T*), hipMemcpyHostToDevice));
	for (int i = 0; i < batchSize; i++)
		cudacall(hipMemcpy(A_dflat + (i * n * n), src[i], n * n * sizeof(T), hipMemcpyHostToDevice));

	T** C = (T**)malloc(batchSize * sizeof(T*));
	T** C_d, * C_dflat;
	cudacall(hipMalloc(&C_d, batchSize * sizeof(T*)));
	cudacall(hipMalloc(&C_dflat, n * n * batchSize * sizeof(T)));
	C[0] = C_dflat;
	for (int i = 1; i < batchSize; i++)
		C[i] = C[i - 1] + (n * n);
	cudacall(hipMemcpy(C_d, C, batchSize * sizeof(T*), hipMemcpyHostToDevice));

	hipDeviceSynchronize();
	QueryPerformanceCounter(&pc_start_inner);

	if (std::is_same<T, double>::value)
		cublascall(hipblasDgetrfBatched(handle, n, (double**)A_d, lda, P, INFO, batchSize));
	else if (std::is_same<T, float>::value)
		cublascall(hipblasSgetrfBatched(handle, n, (float**)A_d, lda, P, INFO, batchSize));

	//cudacall(hipMemcpy(INFOh, INFO, batchSize * sizeof(int), hipMemcpyDeviceToHost));
	//for (int i = 0; i < batchSize; i++)
	//	if (INFOh[i] != 0)
	//	{
	//		fprintf(stderr, "Factorization of matrix %d Failed: Matrix may be singular\n", i);
	//		hipDeviceReset();
	//		exit(EXIT_FAILURE);
	//	}

	//if (std::is_same<T, double>::value)
	//	cublascall(hipblasDgetriBatched(handle, n, (const double**)A_d, lda, P, (double**)C_d, lda, INFO, batchSize));
	//else if (std::is_same<T, float>::value)
	//	cublascall(hipblasSgetriBatched(handle, n, (const float**)A_d, lda, P, (float**)C_d, lda, INFO, batchSize));

	hipDeviceSynchronize();
	QueryPerformanceCounter(&pc_finish_inner);
	pc_diff_inner.QuadPart += pc_finish_inner.QuadPart - pc_start_inner.QuadPart;

	//cudacall(hipMemcpy(INFOh, INFO, batchSize * sizeof(int), hipMemcpyDeviceToHost));
	//for (int i = 0; i < batchSize; i++)
	//	if (INFOh[i] != 0)
	//	{
	//		fprintf(stderr, "Inversion of matrix %d Failed: Matrix may be singular\n", i);
	//		hipDeviceReset();
	//		exit(EXIT_FAILURE);
	//	}
	for (int i = 0; i < batchSize; i++)
		cudacall(hipMemcpy(dst[i], C_dflat + (i * n * n), n * n * sizeof(T), hipMemcpyDeviceToHost));

	hipFree(A_d); hipFree(A_dflat); free(A);
	hipFree(C_d); hipFree(C_dflat); free(C);
	hipFree(P); hipFree(INFO); hipblasDestroy(handle);
	delete[] INFOh;
}

template <class T>
void test_item(const int n, const int mybatch, T null, bool decomponly, LARGE_INTEGER& pc_diff_inner)
{
	T** results = (T**)malloc(mybatch * sizeof(T*));
	for (auto i = 0; i != mybatch; ++i)
	{
		results[i] = (T*)malloc(n * n * sizeof(T));
		if (results[i] == NULL)
			exit(-1);
	}

	T** inputs = (T**)malloc(mybatch * sizeof(T*));
	for (auto i = 0; i != mybatch; ++i)
	{
		inputs[i] = (T*)malloc(n * n * sizeof(T));
		if (inputs[i] == NULL)
			exit(-1);
	}
	//for (auto i = 0; i != mybatch; ++i)
	//	read_1D("A" + std::to_string(n) + ".txt", inputs[i], n * n);
	for (auto i = 0; i != mybatch; ++i)
		setHankelMatrix(inputs[i], n);

	if (!decomponly)
		invert(inputs, results, n, mybatch, pc_diff_inner);
	else
		decomposition_only(inputs, results, n, mybatch, pc_diff_inner);

	//write_1D("iA.txt", results[0], n * n);
	for (auto i = 0; i != mybatch; ++i)
	{
		free(inputs[i]);
		free(results[i]);
	}
	free(inputs);
	free(results);
}

void test_item_onetypeall(const int n, const int mybatch, const std::string& type, bool decomponly, const size_t testnum, const LARGE_INTEGER& pf)
{
	double null_d = 0;
	float null_f = 0;
	LARGE_INTEGER pc_diff_inner;
	ZeroMemory(&pc_diff_inner, sizeof(LARGE_INTEGER));
	if (type == "double")
	{
		for (auto i = 0; i != testnum; ++i)
			test_item(n, mybatch, null_d, decomponly, pc_diff_inner);
	}
	else if (type == "float")
	{
		for (auto i = 0; i != testnum; ++i)
			test_item(n, mybatch, null_f, decomponly, pc_diff_inner);
	}
	else
	{
		std::cout << "Unexpected Type: " << type << std::endl;
		return;
	}
	std::cout << "n = " << n << ", mybatch = " << mybatch << ", Type = " << type << ", DecompOnly = " << decomponly << ", TestNum = " << testnum << ", time = " << std::scientific << (double)(pc_diff_inner.QuadPart) / mybatch / testnum  / pf.QuadPart << "s." << std::endl;
}

int main()
{
	std::cout << "Test: Time comsuming of LU decomposition matrix inversion with CUDA." << std::endl << std::endl;

	int nDevices;
	hipGetDeviceCount(&nDevices);
	for (int i = 0; i < nDevices; i++)
	{
		hipDeviceProp_t prop;
		hipGetDeviceProperties(&prop, i);
		printf("Device Number: %d\n", i);
		printf("  Device name: %s\n", prop.name);
		printf("  Clock Rate (KHz): %d\n",
			prop.clockRate);
		printf("  Memory Clock Rate (KHz): %d\n",
			prop.memoryClockRate);
		printf("  Memory Bus Width (bits): %d\n",
			prop.memoryBusWidth);
		printf("  Peak Memory Bandwidth (GB/s): %f\n\n",
			2.0 * prop.memoryClockRate * (prop.memoryBusWidth / 8) / 1.0e6);
	}

	LARGE_INTEGER pf;
	QueryPerformanceFrequency(&pf);

	std::vector<TestItem> v_ti = LoadTestConfig("TestConfig.ini");

	for (const auto& tmp_ti : v_ti)
		test_item_onetypeall(tmp_ti.n, tmp_ti.mybatch, tmp_ti.type, tmp_ti.decomponly, tmp_ti.testnum, pf);

	return 0;
}